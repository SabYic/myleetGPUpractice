#include "hip/hip_runtime.h"
// 对比：你的原始代码（慢） VS 改进后的代码（快）
// vectorized tiled maxmul GPT改进 没改进应该是load store没合并的问题
///////////////////////////////
// 🔴 原始版本（性能较差）
///////////////////////////////
__global__ void matmul_float2_slow(const float* A, const float* B, float* C, int M, int N, int K) {
    const floatX* A_vec = reinterpret_cast<const floatX*>(A);
    const floatX* B_vec = reinterpret_cast<const floatX*>(B);
    const floatX* C_vec = reinterpret_cast<const floatX*>(C);

    int row_tile = threadIdx.y;
    int col_tile = threadIdx.x;

    int globalRow = blockIdx.y * TS + row_tile;
    int globalCol = blockIdx.x * TS / WIDTH + col_tile;

    __shared__ floatX Asub[TS][TS / WIDTH];
    __shared__ floatX Bsub[TS][TS / WIDTH];

    floatX acc = make_float2(0.f, 0.f);

    int numTiles = ((K + TS / WIDTH - 1) / (TS / WIDTH));

    for (int t = 0; t < numTiles; ++t) {
        int tiledrow = TS * t + row_tile;
        int tiledcol = TS / WIDTH * t + col_tile;

        float valA0 = 0.f, valA1 = 0.f;
        if (globalRow < M) {
            valA0 = (tiledcol * WIDTH < K) ? A[globalRow * K + tiledcol * WIDTH] : 0.0f;
            valA1 = (tiledcol * WIDTH + 1 < K) ? A[globalRow * K + tiledcol * WIDTH + 1] : 0.0f;
        }
        Asub[row_tile][col_tile] = make_float2(valA0, valA1);

        float valB0 = 0.f, valB1 = 0.f;
        if (globalCol * WIDTH < N && tiledrow < K)
            valB0 = B[tiledrow * N + globalCol * WIDTH];
        if (globalCol * WIDTH + 1 < N && tiledrow < K)
            valB1 = B[tiledrow * N + globalCol * WIDTH + 1];
        Bsub[row_tile][col_tile] = make_float2(valB0, valB1);

        __syncthreads();

        for (int k = 0; k < TS / WIDTH; ++k) {
            floatX vecA = Asub[row_tile][k];
            for (int w = 0; w < WIDTH; w++) {
                floatX vecB = Bsub[WIDTH * k + w][col_tile];
                float valA = (w == 0) ? vecA.x : vecA.y;
                acc.x += vecB.x * valA;
                acc.y += vecB.y * valA;
            }
        }
        __syncthreads();
    }

    if (globalRow < M) {
        if (globalCol * WIDTH + 0 < N) C[globalRow * N + globalCol * WIDTH + 0] = acc.x;
        if (globalCol * WIDTH + 1 < N) C[globalRow * N + globalCol * WIDTH + 1] = acc.y;
    }
}


///////////////////////////////
// 🟢 改进版本（性能更好） best performance 418ms(tesla T4) beat 92.4
///////////////////////////////
__global__ void matmul_float2_fast(const float* A, const float* B, float* C, int M, int N, int K) {
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int globalRow = blockIdx.y * TS + ty;
    int globalCol = (blockIdx.x * (TS / WIDTH) + tx) * WIDTH;

    __shared__ floatX Asub[TS][TS / WIDTH];
    __shared__ floatX Bsub[TS][TS / WIDTH];

    floatX acc = make_float2(0.f, 0.f);
    int numTiles = (K + TS - 1) / TS;

    for (int t = 0; t < numTiles; ++t) {
        int tiledRow = t * TS + ty;
        int tiledCol = (t * TS + tx * WIDTH);

        float valA0 = 0.f, valA1 = 0.f;
        if (globalRow < M) {
            if (tiledCol + 0 < K) valA0 = A[globalRow * K + tiledCol + 0];
            if (tiledCol + 1 < K) valA1 = A[globalRow * K + tiledCol + 1];
        }
        Asub[ty][tx] = make_float2(valA0, valA1);

        float valB0 = 0.f, valB1 = 0.f;
        if (tiledRow < K) {
            if (globalCol + 0 < N) valB0 = B[tiledRow * N + globalCol + 0];
            if (globalCol + 1 < N) valB1 = B[tiledRow * N + globalCol + 1];
        }
        Bsub[ty][tx] = make_float2(valB0, valB1);

        __syncthreads();

        for (int k = 0; k < TS / WIDTH; ++k) {
            floatX vecB = Bsub[k][tx];
            for (int w = 0; w < WIDTH; ++w) {
                floatX vecA = Asub[WIDTH * k + w][ty];
                float valB = (w == 0) ? vecB.x : vecB.y;
                acc.x += vecA.x * valB;
                acc.y += vecA.y * valB;
            }
        }

        __syncthreads();
    }

    if (globalRow < M) {
        if (globalCol + 0 < N) C[globalRow * N + globalCol + 0] = acc.x;
        if (globalCol + 1 < N) C[globalRow * N + globalCol + 1] = acc.y;
    }
}
